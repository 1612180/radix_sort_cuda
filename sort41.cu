#include "hip/hip_runtime.h"
#include "sort.h"

//   0 1 2 ... gridDim.x - 1
// 0
// 1
// 2
// ...
// nBins - 1
// hist: 2D array
// convert to 1D array
// (0 1 2 ... gridDim.x - 1)  (of 0) ... (0 1 2 ... gridDim.x - 1) (of nBins -
// 1)
__global__ void computeHist2DKernel41(uint32_t *in, int n, uint32_t *hist,
                                      int nBins, int bitBig) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    int bin = (in[i] >> bitBig) & (nBins - 1);
    atomicAdd(&hist[bin * gridDim.x + blockIdx.x], 1);
  }
}

// in -> out, blockSums
__global__ void scanBlockKernel41(uint32_t *in, int n, uint32_t *out,
                                  uint32_t *blockSums) {
  extern __shared__ uint32_t section[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    section[threadIdx.x] = in[i];
  }
  __syncthreads();

  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    // copy section[threadIdx.x - stride] before changed
    int previous = 0;
    if (stride <= threadIdx.x) {
      previous = section[threadIdx.x - stride];
    }
    __syncthreads();

    section[threadIdx.x] += previous;
    __syncthreads();
  }

  __syncthreads();
  if (i < n) {
    out[i] = section[threadIdx.x];
  }

  if (blockSums != NULL && threadIdx.x == 0) {
    blockSums[blockIdx.x] = section[blockDim.x - 1];
  }
}

__global__ void addScannedBlockSums41(uint32_t *out, int n,
                                      uint32_t *blockSums) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  // skip first block index
  if (i < n && blockIdx.x > 0) {
    out[i] += blockSums[blockIdx.x - 1];
  }
}

__global__ void inclusiveToExclusive41(uint32_t *in, int n, uint32_t *inclusive,
                                       uint32_t *exclusive) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    exclusive[i] = inclusive[i] - in[i];
  }
}

__global__ void convertBinary41(uint32_t *in, int n, uint32_t *inBinary,
                                int nBins, int bitBig, int bitSmall) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    int temp = (in[i] >> bitBig) & (nBins - 1);
    inBinary[i] = (temp >> bitSmall) & 1;
  }
}

__global__ void calculateRankPerBlock41(uint32_t *inBinary, int n,
                                        uint32_t *inBinaryScan,
                                        uint32_t *nZerosPerBlock,
                                        uint32_t *inRankPerBlock) {
  // calculate nZeros for each block
  int lastBlock = blockIdx.x * blockDim.x + blockDim.x - 1;
  int countBlock = blockDim.x;

  // last block is bigger than remain elements
  // example
  // n = 3 => 0 1 2
  // gridSize: 2 -> 2 blocks
  // blockSize: 2 -> 2 threads
  // (0 1) (2)
  // last block only contain (2)
  //
  // count thread in last block is actually < blockDim.x

  if (lastBlock >= n) {
    lastBlock = n - 1;
    countBlock = n - (blockIdx.x * blockDim.x);
  }

  if (threadIdx.x == 0) {
    nZerosPerBlock[blockIdx.x] =
        countBlock - inBinaryScan[lastBlock] - inBinary[lastBlock];
  }
  __syncthreads();

  // calculate rank with nZeros
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    if (inBinary[i] == 0) {
      // threadIdx.x replace for i because we only calculate inside block
      inRankPerBlock[i] = threadIdx.x - inBinaryScan[i];
    } else if (inBinary[i] == 1) {
      inRankPerBlock[i] = nZerosPerBlock[blockIdx.x] + inBinaryScan[i];
    }
  }
}

__global__ void coutingSortPerBlock41(uint32_t *in, int n,
                                      uint32_t *inRankPerBlock, uint32_t *out) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    // because inRankPerBlock[i] is only rank in[i] inside block
    // we need to plus the previous block
    int rank = blockIdx.x * blockDim.x + inRankPerBlock[i];
    out[rank] = in[i];
  }
}

// 1 1 2 2 3 3 4 4 4
// init 1
// 1 1 1 1 1 1 1 1 1
// stride = 1
// 1 2 1 2 1 2 1 2 2
// stride = 2
// 1 2 1 2 1 2 1 2 3
// decrease by 1
// 0 1 0 1 0 1 0 1 2
__global__ void scanBlockKernelWithEqual41(uint32_t *in, int n,
                                           uint32_t *outWithEqual, int nBins,
                                           int bitBig) {
  extern __shared__ uint32_t section[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // copy in to SMEM
  if (i < n) {
    section[threadIdx.x] = 1;
  }
  __syncthreads();

  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    // copy previous section[threadIdx.x - stride] before changed
    int previous = 0;
    if (stride <= threadIdx.x) {
      previous = section[threadIdx.x - stride];
    }
    __syncthreads();

    // only + if equal
    if (stride <= threadIdx.x && i < n) {
      int bin = (in[i] >> bitBig) & (nBins - 1);
      int binPrevious = (in[i - stride] >> bitBig) & (nBins - 1);
      if (bin == binPrevious) {
        section[threadIdx.x] += previous;
      }
    }
    __syncthreads();
  }

  __syncthreads();
  if (i < n) {
    outWithEqual[i] = section[threadIdx.x] - 1;
  }
}

__global__ void coutingSortWithHist41(uint32_t *in, int n, uint32_t *out,
                                      uint32_t *histScan, int nBins, int bitBig,
                                      uint32_t *outWithEqual) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    int bin = (in[i] >> bitBig) & (nBins - 1);
    int rankHist = histScan[bin * gridDim.x + blockIdx.x];

    int rankPerBlock = outWithEqual[i];

    out[rankHist + rankPerBlock] = in[i];
  }
}

void sortBase41(uint32_t *in, int n, uint32_t *out, int nBits,
                int *blockSizes) {
  int nBins = 1 << nBits; // 2^nBits

  uint32_t *tempN = (uint32_t *)malloc(n * sizeof(uint32_t));

  uint32_t *d_in, *d_out;
  CHECK(hipMalloc(&d_in, n * sizeof(uint32_t)));
  CHECK(hipMalloc(&d_out, n * sizeof(uint32_t)));
  CHECK(hipMemcpy(d_in, in, n * sizeof(uint32_t), hipMemcpyHostToDevice));

  // histogram kernel
  int histogramBlockSize = blockSizes[0];
  dim3 histogramGridSize((n - 1) / histogramBlockSize + 1);

  int histSize = histogramGridSize.x * nBins;

  uint32_t *d_hist;
  CHECK(hipMalloc(&d_hist, histSize * sizeof(uint32_t)));

  // scan histogram kernel
  int scanHistogramBlockSize = blockSizes[1];
  dim3 scanHistogramGridSize((histSize - 1) / scanHistogramBlockSize + 1);

  uint32_t *blockSums =
      (uint32_t *)malloc(scanHistogramGridSize.x * sizeof(uint32_t));

  uint32_t *d_histScan, *d_blockSums, *d_histScanExclusive;
  CHECK(hipMalloc(&d_histScan, histSize * sizeof(uint32_t)));
  CHECK(hipMalloc(&d_blockSums, scanHistogramGridSize.x * sizeof(uint32_t)));
  CHECK(hipMalloc(&d_histScanExclusive, histSize * sizeof(uint32_t)));

  // other kernel
  int otherBlockSize = blockSizes[2];
  dim3 otherGridSize((n - 1) / otherBlockSize + 1);

  uint32_t *d_inBinary;
  CHECK(hipMalloc(&d_inBinary, n * sizeof(uint32_t)));

  uint32_t *d_inBinaryScan, *d_inBinaryScanExclusive;
  CHECK(hipMalloc(&d_inBinaryScan, n * sizeof(uint32_t)));
  CHECK(hipMalloc(&d_inBinaryScanExclusive, n * sizeof(uint32_t)));

  uint32_t *d_inRankPerBlock, *d_nZerosPerBlock;
  CHECK(hipMalloc(&d_inRankPerBlock, n * sizeof(uint32_t)));
  CHECK(hipMalloc(&d_nZerosPerBlock, otherGridSize.x * sizeof(uint32_t)));

  uint32_t *d_outWithEqual;
  CHECK(hipMalloc(&d_outWithEqual, n * sizeof(uint32_t)));

  for (int bitBig = 0; bitBig < sizeof(uint32_t) * 8; bitBig += nBits) {
    // d_in -> d_hist
    CHECK(hipMemset(d_hist, 0, histSize * sizeof(uint32_t)));
    computeHist2DKernel41<<<histogramGridSize, histogramBlockSize>>>(
        d_in, n, d_hist, nBins, bitBig);
    CHECK(hipDeviceSynchronize());

    // d_hist -> d_histScan
    scanBlockKernel41<<<scanHistogramGridSize, scanHistogramBlockSize,
                        scanHistogramBlockSize * sizeof(uint32_t)>>>(
        d_hist, histSize, d_histScan, d_blockSums);
    CHECK(hipDeviceSynchronize());

    // scan d_blockSums
    CHECK(hipMemcpy(blockSums, d_blockSums,
                     scanHistogramGridSize.x * sizeof(uint32_t),
                     hipMemcpyDeviceToHost));
    for (int i = 1; i < scanHistogramGridSize.x; i += 1) {
      blockSums[i] += blockSums[i - 1];
    }
    CHECK(hipMemcpy(d_blockSums, blockSums,
                     scanHistogramGridSize.x * sizeof(uint32_t),
                     hipMemcpyHostToDevice));

    // d_histScan + d_blockSums
    addScannedBlockSums41<<<scanHistogramGridSize, scanHistogramBlockSize>>>(
        d_histScan, histSize, d_blockSums);
    CHECK(hipDeviceSynchronize());

    // d_hist, d_histScan -> d_histScanExclusive
    inclusiveToExclusive41<<<scanHistogramGridSize, scanHistogramBlockSize>>>(
        d_hist, histSize, d_histScan, d_histScanExclusive);
    CHECK(hipDeviceSynchronize());

    for (int bitSmall = 0; bitSmall < nBits; bitSmall += 1) {
      // d_in -> d_inBinary
      convertBinary41<<<otherGridSize, otherBlockSize>>>(d_in, n, d_inBinary,
                                                       nBins, bitBig, bitSmall);
      CHECK(hipDeviceSynchronize());

      // d_inBinary -> d_inBinaryScan
      scanBlockKernel41<<<otherGridSize, otherBlockSize,
                          otherBlockSize * sizeof(uint32_t)>>>(
          d_inBinary, n, d_inBinaryScan, NULL);
      CHECK(hipDeviceSynchronize());

      // d_inBinary, d_inBinaryScan -> d_inBinaryScanExclusive
      inclusiveToExclusive41<<<otherGridSize, otherBlockSize>>>(
          d_inBinary, n, d_inBinaryScan, d_inBinaryScanExclusive);
      CHECK(hipDeviceSynchronize());

      // d_inBinary, d_inBinaryScanExclusive -> d_nZerosPerBlock,
      // d_inRankPerBlock
      calculateRankPerBlock41<<<otherGridSize, otherBlockSize>>>(
          d_inBinary, n, d_inBinaryScanExclusive, d_nZerosPerBlock,
          d_inRankPerBlock);
      CHECK(hipDeviceSynchronize());

      coutingSortPerBlock41<<<otherGridSize, otherBlockSize>>>(
          d_in, n, d_inRankPerBlock, d_out);
      CHECK(hipDeviceSynchronize());

      uint32_t *temp = d_in;
      d_in = d_out;
      d_out = temp;
    }

    scanBlockKernelWithEqual41<<<otherGridSize, otherBlockSize,
                                 otherBlockSize * sizeof(uint32_t)>>>(
        d_in, n, d_outWithEqual, nBins, bitBig);
    CHECK(hipDeviceSynchronize());

    coutingSortWithHist41<<<histogramGridSize, histogramBlockSize>>>(
        d_in, n, d_out, d_histScanExclusive, nBins, bitBig, d_outWithEqual);
    CHECK(hipDeviceSynchronize());

    uint32_t *temp = d_in;
    d_in = d_out;
    d_out = temp;
  }

  CHECK(hipMemcpy(out, d_in, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
}